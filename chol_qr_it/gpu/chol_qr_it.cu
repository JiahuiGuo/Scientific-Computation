#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil.h>
#include "hipblas.h"
#include "f2c.h"
#include "clapack.h"
#include "fcntl.h"
#include "cblas.h"
/*=============================================================================

extern "C" int sgemm_(char *, char *, int *, int *, int *, float *, float *, 
                      int *, float *, int *, float *, float *, int *);
extern "C" int sgesvd_(char *, char *, int *, int *, float *, int *, float *, 
                       float *, int *, float *, int *, float *, int *, int *);
extern "C" void sgeqrf_(int*, int*, float*, int*, float*, float*, int*, int*);
extern "C" int scopy_(int *, float*, int *, float*, int *);
extern "C" int strmm_(char*, char *, char*, char *, int *, int *, float *, 
                      float *, int *, float *, int *);
extern "C" int strsm_(char *, char *, char *, char *, int *, int *, 
                      float *, float *, int *, float *, int *);
extern "C" int ssyrk_(char *, char *, int *, int *, float *, float *, 
                      int *, float *, float *, int *);
//=============================================================================
*/
//void chol_qr_it(int m, int n, float *A, int lda, float *R){
void chol_qr_it(integer m, integer n, real *A, integer lda, real *R) {
	integer i = 0, k, j, info, lwork = n*n, n2 = n*n, one = 1;
    real *G, *U, *S, *VT, *vt, *tau, *work;  
    real cn = 200.f, alpha = 1.f, zero = 0.f, mins, maxs;
    
    G    = (float*)malloc(n * n * 4);
    VT   = (float*)malloc(n * n * 4);
    U   = (float*)malloc(n * n * 4);
    S    = (float*)malloc(    n * 4);
    work = (float*)malloc(lwork * 4);
    tau  = (float*)malloc(    n * 4);  

    do {
      i++;

			// G= A'*A
      sgemm_("t", "n", &n, &n, &m, &alpha, A, &m, A, &m, &zero, G, &n);
      //ssyrk_("l", "t", &n, &m, &alpha, A, &m, &zero, G, &n);
      //for(j=0; j<n; j++)
      //  for(k=0; k<j; k++)
      //     G[j*n+k] = G[k*n+j];

			// svd
      sgesvd_("n", "a", &n, &n, G, &n, S, U, &n, VT, &n, work, &lwork, &info);

      mins = 100.f, maxs = 0.f;
      for(k=0; k<n; k++){
        S[k] = sqrt(S[k]);

	if (S[k] < mins)  mins = S[k];
	if (S[k] > maxs)  maxs = S[k];
      }

      for(k=0; k<n;k++){
        vt = VT + k*n;
        for(j=0; j<n; j++)
          vt[j]*=S[j];
      } 
      sgeqrf_(&n, &n, VT, &n, tau, work, &lwork, &info);

      if (i==1)
        scopy_(&n2, VT, &one, R, &one);
      else
        strmm_("l", "u", "n", "n", &n, &n, &alpha, VT, &n, R, &n); // VT contains q and r, R is R

      strsm_("r", "u", "n", "n", &m, &n, &alpha, VT, &n, A, &m);    // A is Q

      if (mins > 0.00001f) 
        cn = maxs/mins;

      fprintf(stderr, "\nIteration %d, cond num = %f \n", i, cn);
    } while (cn > 100.f);

    free(G);
    free(VT);
    free(S);
    free(work);
    free(tau);
}

//=============================================================================

//void chol_qr_it_GPU(int m, int n, float *d_A, int lda, float *G, float *R, 
  //               float *work, int lwork){
void chol_qr_it_GPU(integer m, integer n, real *d_A, integer lda, real *G, real *R, 
                 real *work, integer lwork){
		integer i = 0, k, j, info, n2 = n*n, one = 1;
    real *U, *S, *VT, *d_VT, *vt, *tau, *h_G;
    real cn = 200.f, alpha = 1.f, zero = 0.f, mins, maxs;

    h_G  = (float*)malloc(n * n * 4);
    VT   = (float*)malloc(n * n * 4);
    U   = (float*)malloc(n * n * 4);
    cublasAlloc(n * n * 4, 1, (void **)&d_VT);
    S    = (float*)malloc(    n * 4);
    tau  = (float*)malloc(    n * 4);

    do {
      i++;
      hipblasSgemm( 't', 'n', n, n, m, alpha, d_A, m, d_A, m, zero, G, n);
      hipblasGetVector(n*n*4, 1, G, 1, h_G, 1);

      sgesvd_("n", "a", &n, &n, h_G, &n, S, U, &n, VT, &n, work, &lwork, &info);

      mins = 100.f, maxs = 0.f;
      for(k=0; k<n; k++){
        S[k] = sqrt(S[k]);

        if (S[k] < mins)  mins = S[k];
        if (S[k] > maxs)  maxs = S[k];
      }

      for(k=0; k<n;k++){
        vt = VT + k*n;
        for(j=0; j<n; j++)
          vt[j]*=S[j];
      }
      sgeqrf_(&n, &n, VT, &n, tau, work, &lwork, &info);

      if (i==1)
        scopy_(&n2, VT, &one, R, &one);
      else
        strmm_("l", "u", "n", "n", &n, &n, &alpha, VT, &n, R, &n);

      hipblasSetVector(n*n*4, 1, VT, 1, d_VT, 1);
      hipblasStrsm('r', 'u', 'n', 'n', m, n, alpha, d_VT, n, d_A, m);

      if (mins > 0.00001f)
        cn = maxs/mins;
      
      fprintf(stderr, "\nIteration %d, cond num = %f \n", i, cn);
    } while (cn > 100.f);

    free(VT);
    free(S);
    free(tau);
}

//=============================================================================
